#include "hip/hip_runtime.h"
#include "cuda_classifier_ops.h" // Includi l'header che dichiara le funzioni

#include <iostream> // Per std::cerr

// **************************** KERNEL CUDA ****************************
// Kernel per calcolare il prodotto elemento per elemento
__global__ void calculatePartialScoreKernel(const float* dev_imageData, const float* dev_weights, float* dev_partialSums, int num_elements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < num_elements) {
        dev_partialSums[idx] = dev_imageData[idx] * dev_weights[idx];
    }
}

// Kernel per la riduzione parallela (somma dei risultati parziali)
__global__ void reduceSumKernel(float* dev_partialSums, int num_elements, float* dev_finalSum) {
    extern __shared__ float sdata[]; 

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + tid; // Per accedere agli elementi in modo coalescente

    sdata[tid] = 0; 
    if (i < num_elements) {
        sdata[tid] += dev_partialSums[i];
    }
    if (i + blockDim.x < num_elements) { 
        sdata[tid] += dev_partialSums[i + blockDim.x];
    }
    __syncthreads(); 

    for (unsigned int s = blockDim.x / 2; s > 0; s /= 2) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(dev_finalSum, sdata[0]); 
    }
}
// ********************************************************************************

__global__ void classifyBatchKernel(
    const float* dev_images,
    const float* dev_weights,
    float* dev_scores,
    int num_pixels_per_image,
    int num_images
) {
    int image_id = blockIdx.x;
    int tid = threadIdx.x;

    if (image_id >= num_images) return;

    extern __shared__ float sdata[];

    float sum = 0.0f;
    for (int i = tid; i < num_pixels_per_image; i += blockDim.x) {
        int idx = image_id * num_pixels_per_image + i;
        sum += dev_images[idx] * dev_weights[i];
    }

    sdata[tid] = sum;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        dev_scores[image_id] = sdata[0];
    }
}


// Implementazione delle funzioni dichiarate nell'header
void initCudaWeights(const std::vector<float>& host_weights, float** dev_weights, int* num_weights_ptr) {
    *num_weights_ptr = host_weights.size();
    CUDA_CHECK(hipMalloc((void**)dev_weights, host_weights.size() * sizeof(float)));
    CUDA_CHECK(hipMemcpy(*dev_weights, host_weights.data(), host_weights.size() * sizeof(float), hipMemcpyHostToDevice));
}

void freeCudaWeights(float* dev_weights) {
    if (dev_weights) {
        CUDA_CHECK(hipFree(dev_weights));
    }
}

float calculateScoreOnGPU(const std::vector<float>& image_data_host, const float* dev_weights, int num_weights) {
    int num_pixels = image_data_host.size();
    
    // Verifica che la dimensione dell'immagine corrisponda alla dimensione dei pesi
    if (num_pixels != num_weights) {
        std::cerr << "Errore: Dimensione dell'immagine (" << num_pixels << ") non corrisponde alla dimensione dei pesi (" << num_weights << ")." << std::endl;
        exit(EXIT_FAILURE);
    }

    // Allocazione memoria su Device (GPU) per i dati dell'immagine e i risultati intermedi
    float* dev_imageData;
    float* dev_partialSums;
    float* dev_finalScore;
    
    CUDA_CHECK(hipMalloc((void**)&dev_imageData, num_pixels * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&dev_partialSums, num_pixels * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&dev_finalScore, sizeof(float)));
    
    // Inizializza il risultato finale a zero sulla GPU
    CUDA_CHECK(hipMemset(dev_finalScore, 0, sizeof(float)));

    // Copia i dati dell'immagine da Host a Device
    CUDA_CHECK(hipMemcpy(dev_imageData, image_data_host.data(), num_pixels * sizeof(float), hipMemcpyHostToDevice));

    // Configurazione del lancio del Kernel per i prodotti parziali
    int threadsPerBlock = 256; 
    int blocksPerGrid = (num_pixels + threadsPerBlock - 1) / threadsPerBlock; 

    // Lancia il Kernel per i prodotti parziali
    calculatePartialScoreKernel<<<blocksPerGrid, threadsPerBlock>>>(dev_imageData, dev_weights, dev_partialSums, num_pixels);
    CUDA_CHECK(hipDeviceSynchronize()); // Sincronizza per catturare errori

    // Lancia il Kernel per la riduzione finale
    // Per la riduzione, usiamo un solo blocco e una shared memory della dimensione di threadsPerBlock
    // Si noti che la dimensione della shared memory è passata come terzo argomento a <<< >>>
    reduceSumKernel<<<1, threadsPerBlock, threadsPerBlock * sizeof(float)>>>(dev_partialSums, num_pixels, dev_finalScore);
    CUDA_CHECK(hipDeviceSynchronize()); 

    // Copia il risultato finale da Device a Host
    float final_score_host;
    CUDA_CHECK(hipMemcpy(&final_score_host, dev_finalScore, sizeof(float), hipMemcpyDeviceToHost));

    // Liberare la memoria GPU allocata per questa immagine
    CUDA_CHECK(hipFree(dev_imageData));
    CUDA_CHECK(hipFree(dev_partialSums));
    CUDA_CHECK(hipFree(dev_finalScore));
    
    return final_score_host;
}

void classifyBatchOnGPU(
    const std::vector<std::vector<float>>& images,
    const float* dev_weights,
    int num_weights,
    std::vector<float>& out_scores
) {
    int num_images = images.size();
    int num_pixels_per_image = num_weights;

    // Flatten all images in un unico array
    std::vector<float> host_images_flat(num_images * num_pixels_per_image);
    for (int i = 0; i < num_images; ++i) {
        std::copy(images[i].begin(), images[i].end(),
                  host_images_flat.begin() + i * num_pixels_per_image);
    }

    // Alloca device memory
    float* dev_images;
    float* dev_scores;

    CUDA_CHECK(hipMalloc(&dev_images, host_images_flat.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&dev_scores, num_images * sizeof(float)));

    CUDA_CHECK(hipMemcpy(dev_images, host_images_flat.data(),
                          host_images_flat.size() * sizeof(float), hipMemcpyHostToDevice));

    // Configura kernel
    dim3 grid(num_images);
    dim3 block(256);
    size_t sharedMem = 256 * sizeof(float);

    classifyBatchKernel<<<grid, block, sharedMem>>>(
        dev_images, dev_weights, dev_scores,
        num_pixels_per_image, num_images
    );
    CUDA_CHECK(hipDeviceSynchronize());

    // Copia risultati
    out_scores.resize(num_images);
    CUDA_CHECK(hipMemcpy(out_scores.data(), dev_scores,
                          num_images * sizeof(float), hipMemcpyDeviceToHost));

    hipFree(dev_images);
    hipFree(dev_scores);
}
