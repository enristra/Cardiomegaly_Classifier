#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <string>
#include <filesystem>
#include <algorithm>
#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include "cnpy.h"

namespace fs = std::filesystem;

#define CHECK_CUDA(call) do { \
    hipError_t err = (call);  \
    if (err != hipSuccess) {  \
        std::cerr << "CUDA error " << hipGetErrorString(err) \
                  << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        std::exit(1); \
    } \
} while(0)

static constexpr int H = 224;
static constexpr int W = 224;
static constexpr int P = H*W; // 50176

// Kernel: un blocco per immagine, riduzione in shared memory
__global__ void dot_per_image_kernel(const float* __restrict__ images,
                                     const float* __restrict__ weights,
                                     float* __restrict__ scores,
                                     int pixels_per_img) {
    extern __shared__ float sdata[]; // dimensione = blockDim.x
    const int tid = threadIdx.x;
    const int img_idx = blockIdx.x;

    const float* img = images + img_idx * pixels_per_img;

    float sum = 0.0f;
    // ogni thread attraversa i pixel a stride
    for (int i = tid; i < pixels_per_img; i += blockDim.x) {
        // letture coalesced su img; weights è uguale per tutti
        sum += img[i] * weights[i];
    }

    sdata[tid] = sum;
    __syncthreads();

    // riduzione in shared
    for (int s = blockDim.x/2; s > 0; s >>= 1) {
        if (tid < s) sdata[tid] += sdata[tid + s];
        __syncthreads();
    }

    if (tid == 0) {
        scores[img_idx] = sdata[0];
    }
}

// -------- Utils I/O --------
static std::vector<float> load_weights_or_die(const std::string& path) {
    std::cout << "Carico pesi: " << path << std::endl;
    cnpy::NpyArray arr = cnpy::npy_load(path);
    std::cout << "Pesi caricati. Shape: " << arr.shape[0] << "x" << arr.shape[1] << std::endl;

    if (arr.shape.size() != 2 || arr.shape[0] != H || arr.shape[1] != W)
        throw std::runtime_error("weights.npy deve essere 224x224");
    std::vector<float> w;
    if (arr.word_size == sizeof(float)) {
        float* p = arr.data<float>();
        w.assign(p, p + arr.num_vals);
    } else if (arr.word_size == sizeof(double)) {
        double* p = arr.data<double>();
        w.resize(arr.num_vals);
        for (size_t i=0;i<arr.num_vals;++i) w[i] = static_cast<float>(p[i]);
    } else {
        throw std::runtime_error("Tipo pesi non supportato");
    }
    // (opzionale) normalizza a somma 1 per avere media pesata
    double s=0.0; for (float v: w) s += v;
    if (s > 0) for (auto& v: w) v = static_cast<float>(v / s);
    return w;
}

static void save_scores_npy(const std::string& path, const std::vector<float>& scores) {
    std::vector<size_t> shape = { scores.size() };
    cnpy::npy_save(path, scores.data(), shape, "w");
    std::cout << "Salvato: " << path << std::endl;
}

// Carica batch N×P (float32) se esiste, altrimenti legge directory con image_*.npy (uint8 o float)
static std::vector<float> load_images_batch_or_dir(const std::string& batch_path,
                                                   const std::string& images_dir,
                                                   size_t& N_out) {
    // 1) Tenta BATCH se esiste
    if (!batch_path.empty() && fs::exists(batch_path)) {
        std::cout << "[IMG] Provo batch: " << batch_path << std::endl << std::flush;
        cnpy::NpyArray arr = cnpy::npy_load(batch_path);
        std::cout << "[IMG] Batch caricato. dims=" << arr.shape.size() << " [";
        for (size_t i=0;i<arr.shape.size();++i){ std::cout << arr.shape[i] << (i+1<arr.shape.size()?'x':']'); }
        std::cout << "] word_size=" << arr.word_size << std::endl << std::flush;

        // Supporta sia (N, P) che (N, 224, 224)
        if (arr.shape.size() == 2 && arr.shape[1] == P) {
            if (arr.word_size != sizeof(float))
                throw std::runtime_error("images_batch.npy (N x 50176) deve essere float32");
            N_out = arr.shape[0];
            const float* p = arr.data<float>();
            return std::vector<float>(p, p + N_out*P);
        } else if (arr.shape.size() == 3 && arr.shape[1] == H && arr.shape[2] == W) {
            // Converte (N,224,224) -> (N, P)
            N_out = arr.shape[0];
            std::vector<float> out; out.reserve(N_out*P);
            if (arr.word_size == sizeof(uint8_t)) {
                const uint8_t* p = arr.data<uint8_t>();
                for (size_t n=0;n<N_out;++n) {
                    const uint8_t* img = p + n*P;
                    for (int i=0;i<P;++i) out.push_back(img[i] / 255.0f);
                }
            } else if (arr.word_size == sizeof(float)) {
                const float* p = arr.data<float>();
                for (size_t n=0;n<N_out;++n) {
                    const float* img = p + n*P;
                    out.insert(out.end(), img, img + P);
                }
            } else {
                throw std::runtime_error("images_batch.npy (N,224,224) deve essere uint8 o float32");
            }
            std::cout << "[IMG] Batch convertito a (N,P). N=" << N_out << std::endl << std::flush;
            return out;
        } else {
            throw std::runtime_error("images_batch.npy shape inattesa (supporto: (N,50176) o (N,224,224))");
        }
    }

    // 2) Fallback: DIRECTORY
    std::cout << "[IMG] Batch non trovato (" << batch_path << "), provo directory: " 
              << images_dir << std::endl << std::flush;
    if (!fs::exists(images_dir)) {
        throw std::runtime_error("Directory immagini non esiste: " + images_dir);
    }

    std::vector<fs::path> files;
    for (const auto& e : fs::directory_iterator(images_dir)) {
        if (e.path().extension() == ".npy" &&
            e.path().filename().string().rfind("image_", 0) == 0) {
            files.push_back(e.path());
        }
    }
    std::sort(files.begin(), files.end());
    if (files.empty()) throw std::runtime_error("Nessuna npy trovata in " + images_dir);

    std::cout << "[IMG] Trovati " << files.size() << " file .npy nella dir." << std::endl << std::flush;

    std::vector<float> all;
    all.reserve(files.size()*P);

    size_t cnt=0;
    for (auto& f : files) {
        if ((++cnt % 500) == 0) {
            std::cout << "[IMG] Caricate " << cnt << " immagini..." << std::endl << std::flush;
        }
        cnpy::NpyArray a = cnpy::npy_load(f.string());
        if (a.shape.size() == 2 && a.shape[0]==H && a.shape[1]==W) {
            if (a.word_size == sizeof(uint8_t)) {
                const uint8_t* p = a.data<uint8_t>();
                for (int i=0;i<P;++i) all.push_back(p[i] / 255.0f);
            } else if (a.word_size == sizeof(float)) {
                const float* p = a.data<float>();
                all.insert(all.end(), p, p+P);
            } else {
                throw std::runtime_error("Formato immagine non supportato (224x224)");
            }
        } else if (a.shape.size()==1 && a.shape[0]==P) {
            if (a.word_size != sizeof(float))
                throw std::runtime_error("Immagine flatten deve essere float32");
            const float* p = a.data<float>();
            all.insert(all.end(), p, p+P);
        } else {
            throw std::runtime_error("Dimensione inattesa in " + f.string());
        }
    }
    N_out = files.size();
    std::cout << "[IMG] Caricate " << N_out << " immagini totali." << std::endl << std::flush;
    return all;
}


// (opzionale) sequenziale CPU per confronto tempi/correttezza
static std::vector<float> cpu_dot_scores(const std::vector<float>& imgs,
                                         const std::vector<float>& w,
                                         size_t N) {
    std::vector<float> s(N, 0.0f);
    for (size_t n=0;n<N;++n) {
        const float* img = imgs.data() + n*P;
        double acc=0.0;
        for (int i=0;i<P;++i) acc += img[i]*w[i];
        s[n] = static_cast<float>(acc);
    }
    return s;
}

// -------- MAIN --------
int main(int argc, char** argv) {
    try {
        // Path di default
        std::string weights_path = "../data/weights/static_weights_224x224.npy";
        std::string images_batch = "../data/images_batch_224x224.npy"; // se non esiste, usa directory
        std::string images_dir   = "../data/ChestMNIST_Images";
        std::string out_scores   = "../results/scores_gpu.npy";
        bool run_cpu_baseline = true;
        int block_size = 256;

        // parse argomenti semplici
        for (int i=1;i<argc;++i) {
            if (!std::strcmp(argv[i], "--weights") && i+1<argc) weights_path = argv[++i];
            else if (!std::strcmp(argv[i], "--batch") && i+1<argc) images_batch = argv[++i];
            else if (!std::strcmp(argv[i], "--images") && i+1<argc) images_dir = argv[++i];
            else if (!std::strcmp(argv[i], "--out") && i+1<argc) out_scores = argv[++i];
            else if (!std::strcmp(argv[i], "--bs") && i+1<argc) block_size = std::stoi(argv[++i]);
            else if (!std::strcmp(argv[i], "--no-cpu")) run_cpu_baseline = false;
        }

        // crea cartella results
        try { fs::create_directories(fs::path(out_scores).parent_path()); } catch(...) {}

        // Carica pesi (normalizzati a somma = 1)
        auto weights = load_weights_or_die(weights_path);

        // Carica immagini (batch o dir)
        size_t N = 0;
        auto images = load_images_batch_or_dir(images_batch, images_dir, N);
        if (images.size() != N*P) throw std::runtime_error("Dimensioni immagini non coerenti");

        std::cout << "N = " << N << ", P = " << P << ", block_size = " << block_size << std::endl;

        // --- CPU baseline (opzionale) ---
        std::vector<float> cpu_scores;
        double cpu_ms = 0.0;
        if (run_cpu_baseline) {
            auto t0 = std::chrono::high_resolution_clock::now();
            cpu_scores = cpu_dot_scores(images, weights, N);
            auto t1 = std::chrono::high_resolution_clock::now();
            cpu_ms = std::chrono::duration<double, std::milli>(t1 - t0).count();
            std::cout << "[CPU] dot total: " << cpu_ms << " ms" << std::endl;
        }

        // --- GPU alloc ---
        float *d_images=nullptr, *d_weights=nullptr, *d_scores=nullptr;
        CHECK_CUDA(hipMalloc((void**)&d_images,  N*P*sizeof(float)));
        CHECK_CUDA(hipMalloc((void**)&d_weights, P*sizeof(float)));
        CHECK_CUDA(hipMalloc((void**)&d_scores,  N*sizeof(float)));

        // copy H2D
        CHECK_CUDA(hipMemcpy(d_images,  images.data(),  N*P*sizeof(float), hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(d_weights, weights.data(), P*sizeof(float),   hipMemcpyHostToDevice));

        // --- GPU kernel ---
        dim3 grid(N);
        dim3 block(block_size);
        size_t shmem = block_size * sizeof(float);

        hipEvent_t ev_start, ev_stop;
        CHECK_CUDA(hipEventCreate(&ev_start));
        CHECK_CUDA(hipEventCreate(&ev_stop));

        CHECK_CUDA(hipEventRecord(ev_start));
        dot_per_image_kernel<<<grid, block, shmem>>>(d_images, d_weights, d_scores, P);
        CHECK_CUDA(hipEventRecord(ev_stop));
        CHECK_CUDA(hipEventSynchronize(ev_stop));

        float gpu_ms = 0.0f;
        CHECK_CUDA(hipEventElapsedTime(&gpu_ms, ev_start, ev_stop));
        std::cout << "[GPU] kernel time: " << gpu_ms << " ms" << std::endl;

        // copy D2H
        std::vector<float> scores(N);
        CHECK_CUDA(hipMemcpy(scores.data(), d_scores, N*sizeof(float), hipMemcpyDeviceToHost));

        // confronta con CPU (se fatto)
        if (run_cpu_baseline) {
            double mae = 0.0, maxae = 0.0;
            for (size_t i=0;i<N;++i) {
                double ae = std::abs((double)scores[i] - (double)cpu_scores[i]);
                mae += ae;
                if (ae > maxae) maxae = ae;
            }
            mae /= N;
            std::cout << "Confronto CPU vs GPU -> MAE: " << mae << "  maxAE: " << maxae << std::endl;
        }

        // salva risultati GPU
        save_scores_npy(out_scores, scores);

        // cleanup
        CHECK_CUDA(hipFree(d_images));
        CHECK_CUDA(hipFree(d_weights));
        CHECK_CUDA(hipFree(d_scores));
        CHECK_CUDA(hipEventDestroy(ev_start));
        CHECK_CUDA(hipEventDestroy(ev_stop));

        return 0;
    } catch (const std::exception& e) {
        std::cerr << "Errore: " << e.what() << std::endl;
        return 1;
    }
}
